
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

//Adapted from Nvidia
__device__ void sum_reduction(int *data, int *out) {
    unsigned int id = threadIdx.x;
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
	if (id < s) {
	    data[id] += data[id + s];
	}
	__syncthreads();
    }
    if (id == 0) out[0] = data[0];
    __syncthreads();
}

/*
__device__ void reduce_sum(float *data, float *temp, float* out) {
    unsigned int tid = threadIdx.x;
    temp[tid] = data[tid];
    __syncthreads();
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
	if (tid < s) {
	    sdata[tid] += sdata[tid + s];
	}
	__syncthreads();
    }
    if (tid == 0) {
	out[0] = temp[0];
    }
}
*/

__device__ void max_func(short *data, short *temp, short *out) {
    unsigned int tid = threadIdx.x;
    temp[tid] = data[tid];
    __syncthreads();
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
	if (tid < s) {
	    temp[tid] = ((temp[tid+s] > temp[tid]) ? temp[tid+s] : temp[tid]);
	}
	__syncthreads();
    }
    if (tid == 0) {
	out[0] = temp[0];
    }
}

__device__ void max_func_special(short *data, short *temp, short *out, int stride, int offset) {
    unsigned int tid = threadIdx.x;
    temp[tid] = data[tid];
    __syncthreads();
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
	if ((tid-offset) < s && (tid-offset) > 0 && (tid-offset) % stride == 0) {
	    temp[tid] = ((temp[tid+s] > temp[tid]) ? temp[tid+s] : temp[tid]);
	}
	__syncthreads();
    }
    if (tid == 0) {
	out[0] = temp[0];
    }
}

__device__ void min_func(short *data, short *temp, short *out) {
    unsigned int tid = threadIdx.x;
    temp[tid] = data[tid];
    __syncthreads();
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
	if (tid < s) {
	    temp[tid] = ((temp[tid+s] < temp[tid]) ? temp[tid+s] : temp[tid]);
	}
	__syncthreads();
    }
    if (tid == 0) {
	out[0] = temp[0];
    }
}

__device__ void block_scan(int *data) {
	unsigned int tid = threadIdx.x;
	for (unsigned int d = 1; d<blockDim.x; d<<=1) {
		if ((tid + 1) % (d<<1) == 0) {
			data[tid] = data[tid] + data[tid - d];
		}
		__syncthreads();
	}

	if (tid==blockDim.x-1) {
		data[tid] = 0;
	}
	__syncthreads();
	int tmp;
	for (unsigned int d = blockDim.x>>1; d >= 1; d>>=1) {
		if ((tid + 1) % (d<<1) == 0) {
			tmp = data[tid - d];
			data[tid - d] = data[tid];
			data[tid] = tmp + data[tid];
		}
		__syncthreads();
	}
}

__device__ void block_scan(float *data) {
	unsigned int tid = threadIdx.x;
	for (unsigned int d = 1; d<blockDim.x; d<<=1) {
		if ((tid + 1) % (d<<1) == 0) {
			data[tid] = data[tid] + data[tid - d];
		}
		__syncthreads();
	}

	if (tid==blockDim.x-1) {
		data[tid] = 0;
	}
	__syncthreads();
	float tmp;
	for (unsigned int d = blockDim.x>>1; d >= 1; d>>=1) {
		if ((tid + 1) % (d<<1) == 0) {
			tmp = data[tid - d];
			data[tid - d] = data[tid];
			data[tid] = tmp + data[tid];
		}
		__syncthreads();
	}
}

__device__ void radix_sort(int *data, int *temp1, int *temp2) {
    unsigned int tid = threadIdx.x;
    unsigned int total = 0;
    unsigned int b = 0;
    for (unsigned int k=0; k<sizeof(int)*8; ++k) {
	b = (data[tid] & (1 << k)) == 0; //Actually opposite of bit
	temp1[tid] = b;
	temp2[tid] = b;
	__syncthreads();
	block_scan(temp1);
	total = temp1[blockDim.x-1] + temp2[blockDim.x-1];
	temp2[tid] = tid - temp1[tid] + total;
	temp1[tid] = b ? temp1[tid] : temp2[tid]; //Inverse of nvidia radix, account for b being !bit
	int tmp = data[tid];
	__syncthreads();
	data[temp1[tid]] = tmp;
	__syncthreads();
    }
}

__device__ void radix_sort_by_key(int *keys, int *data, int *temp1, int *temp2) {
    unsigned int tid = threadIdx.x;
    unsigned int total = 0;
    unsigned int b = 0;
    for (unsigned int k=0; k<sizeof(int)*8; ++k) {
	b = (keys[tid] & (1 << k)) == 0; //Actually opposite of bit
	temp1[tid] = b;
	temp2[tid] = b;
	__syncthreads();
	block_scan(temp1);
	total = temp1[blockDim.x-1] + temp2[blockDim.x-1];
	temp2[tid] = tid - temp1[tid] + total;
	temp1[tid] = b ? temp1[tid] : temp2[tid]; //Inverse of nvidia radix, account for b being !bit
	int tmp_data = data[tid];
	int tmp_key = keys[tid];
	__syncthreads();
	data[temp1[tid]] = tmp_data;
	keys[temp1[tid]] = tmp_key;
	__syncthreads();
    }
}

__global__ void test_kernel(int *test_int_data, short *test_short_data) {
    //Need blockdim of 256, one block
    __shared__ int test_int[1024];
    __shared__ int temp_int_1[1024];
    __shared__ int temp_int_2[1024];
    __shared__ short test_short[1024];
    __shared__ short temp_short[1024];
    __shared__ short out[1];
    unsigned int tid = threadIdx.x;
    test_int[tid] = 1024 - tid;
    temp_int_1[tid] = 1024 - tid;
    test_short[tid] = 1024 - tid;
    __syncthreads();

    if (tid == 0) printf("Running test kernel\n");
    max_func(test_short, temp_short, out);
    if (tid == 0) printf("Max: %d\n", out[0]);
    min_func(test_short, temp_short, out);
    if (tid == 0) printf("Min: %d\n", out[0]);
    block_scan(temp_int_1);
    if (tid == 0) {
	printf("Block scan: [");
	for (unsigned int i=0; i<1024; ++i) {
	    printf("%d, ", temp_int_1[i]);
	}
	printf("]\n");
    }
    __syncthreads();
    temp_int_1[tid] = 1024 - tid;
    block_scan(temp_int_1);
    if (tid == 0) {
	printf("Block scan: [");
	for (unsigned int i=0; i<1024; ++i) {
	    printf("%d, ", temp_int_1[i]);
	}
	printf("]\n");
    }
    __syncthreads();
    radix_sort(test_int, temp_int_1, temp_int_2);
    if (tid == 0) {
	printf("Radix Sort: [");
	for (unsigned int i=0; i<1024; ++i) {
	    printf("%d, ", test_int[i]);
	}
	printf("]\n");
    }
}

