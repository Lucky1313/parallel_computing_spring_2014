#include "hip/hip_runtime.h"



#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#include "layout.h"

using namespace std;

#define TILE_WIDTH 8
#define POP_SIZE 32
#define MIGRATION_FREQ 0

//Data is as follows:
//0 - size of each copy of data
//Node data (all are pairs of x, y):
//PWR
//GND
//INPUTS
//OUTPUTS
//TERMINALS - Always D-G-S
__constant__ short node_layout[32000];

__global__ void rand_setup_kernel(hiprandState *state) {
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    hiprand_init(7+id, id, 0, &state[id]);
}

__global__ void ga_populate_kernel(hiprandState *state, short *pop_mem) {
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    hiprandState local_state = state[id];
    float rand_float;
    int rand;

    int mem_offset = id * node_layout[0] * 2;
    //printf("Memory offset: %d, for thread: %d. Layout size: %d, %d\n", mem_offset, id, node_layout[0], node_layout[1]);
    for (unsigned int i=0; i<node_layout[0] * 2; ++i) {
	rand_float = hiprand_uniform(&local_state) * 100; //TODO Depend on num transistors
	rand = (int) rand_float;
	pop_mem[mem_offset+i] = rand;
    }
    state[id] = local_state;
}

__global__ void ga_fitness_kernel(short *pop_mem, float *fit_mem) {
    __shared__ float fit_scores[TILE_WIDTH];
    int id = blockIdx.x*blockDim.x + threadIdx.x;

    int mem_offset = id * node_layout[1];
}

int term_pos(Terminal *term, int offset_data, int offset_in, int offset_out) {
    switch(term->type) {
    case 'D':
	return 2 + offset_data + offset_in + offset_out + (term->num - 1) * 3; //Order D-G-S
    case 'G':
	return 3 + offset_data + offset_in + offset_out + (term->num - 1) * 3;
    case 'S':
	return 4 + offset_data + offset_in + offset_out + (term->num - 1) * 3;
    case 'P':
	return offset_data; //Always first
    case 'Z':
	return offset_data + 1; //Always second
    case 'I':
	return 1 + offset_data + term->num; //Directly follows ground, term num starts at 1, so only offset 1
    case 'O':
	return 1 + offset_data + offset_in + term->num; // Follows inputs
    default:
	cout << "Improper terminal" << endl;
	return -1;
    }
}

void create_node_data_array(Layout *main_layout, short *node_data, int offset_data, int offset_in, int offset_out) {
    Node* node;
    node_data[offset_data] = main_layout->nodes_size();
    int node_offset = 1 + offset_data + main_layout->nodes_size();
    for (unsigned int i=0; i<main_layout->nodes_size(); ++i) {
	node = main_layout->get_node(i);
	node_data[i+1+offset_data] = node->terms.size();
	for (unsigned int j=0; j<node->terms.size(); ++j) {
	    node_data[node_offset] = term_pos(node->terms[j], offset_data, offset_in, offset_out);
	    ++node_offset;
	}
    }
}

void launch_ga(Layout *main_layout) {
    //Specify block size
    const dim3 block_size(TILE_WIDTH);
    //Assume POP_SIZE is multiple of block size
    const dim3 num_blocks(POP_SIZE/block_size.x);

    int num_terminals = (main_layout->trans_size()) * 3;
    int offset_in = main_layout->in_size();
    int offset_out = main_layout->out_size();
    int trans_offset = offset_in + offset_out + 2;
    short per_copy_size = (trans_offset + num_terminals) * 2 * sizeof(short);
    int thread_count = num_blocks.x * num_blocks.y * block_size.x * block_size.y;
    int total_mem = thread_count * per_copy_size;

    cout << "Read in Layout. " << endl;
    cout << "Offset: " << trans_offset << endl;
    cout << "Number of transistor terminals: " << num_terminals << endl;


    cout << "\n\nBlock Allocation:\nDefined Tile size: " << TILE_WIDTH << endl;
    cout << "Defined population size: " << POP_SIZE << endl;
    cout << "Number of blocks: " << num_blocks.x * num_blocks.y << endl;
    cout << "Number of threads per block: " << block_size.x * block_size.y<< endl;
    cout << "Total number of threads: " << thread_count << endl;

    //Allocate memory
    cout << "\n\nMemory:\nSize of short: " << sizeof(short) << " bytes" << endl;
    cout << "Size of layout copy: " << per_copy_size << " bytes" << endl;
    cout << "Total layout memory use: " << total_mem << " bytes" << endl;

    int size = 0;
    Node* node;
    for (unsigned int i=0; i<main_layout->nodes_size(); ++i) {
	node = main_layout->get_node(i);
	size += node->terms.size();
    }

    int offset_data = 2;
    int node_mem_size = 1 + main_layout->nodes_size() + size + offset_data;
    short node_data[node_mem_size];

    node_data[0] = (trans_offset + num_terminals);
    node_data[1] = per_copy_size;

    cout << "Node size: " << node_mem_size << " numbers" << endl;
    cout << "Node memory use: " << node_mem_size * sizeof(short) << " bytes" << endl;

    create_node_data_array(main_layout, node_data, offset_data, offset_in, offset_out);

    //Random number generation setup
    hiprandState *rand_states = 0;
    hipMalloc((void **)&rand_states, thread_count * sizeof(hiprandState));
    rand_setup_kernel<<<num_blocks, block_size>>>(rand_states);

    short *pop_mem = 0;
    //short *node_mem = 0;
    float *fit_mem = 0;
    short *run_num = 0;

    hipMalloc((void**)&pop_mem, total_mem);
    //hipMalloc((void**)&node_mem, node_mem_size * sizeof(short));
    hipMalloc((void**)&fit_mem, thread_count * sizeof(float));
    hipMalloc((void**)&run_num, sizeof(short));

    //hipMemcpy(node_mem, node_data, node_mem_size * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(node_layout), node_data, node_mem_size * sizeof(short));
    hipMemset(run_num, 0, sizeof(short));

    ga_populate_kernel<<<num_blocks, block_size>>>(rand_states, pop_mem);

    ga_fitness_kernel<<<num_blocks, block_size>>>(pop_mem, fit_mem);

    short *host_pop = 0;
    host_pop = (short*)malloc(total_mem);

    hipMemcpy(host_pop, pop_mem, total_mem, hipMemcpyDeviceToHost);


    cout << "Population: " << endl;
    for (unsigned int i=0; i<(trans_offset + num_terminals) * POP_SIZE; ++i) {
	cout << "(" << host_pop[i*2] << ", " << host_pop[i*2+1] << ") ";
	if ((i + 1) % (trans_offset + num_terminals)  == 0) {
	    cout << endl << endl;
	}
	}


    hipFree(pop_mem);
    hipFree(fit_mem);
    hipFree(run_num);
}

