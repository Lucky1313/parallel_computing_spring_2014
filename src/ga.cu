#include "hip/hip_runtime.h"



#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#include "layout.h"

using namespace std;

#define TILE_WIDTH 8
#define POP_SIZE 32
#define MIGRATION_FREQ 0
#define DISTANCE_WEIGHT 1
#define ANGLE_WEIGHT 1
#define LEFT_WEIGHT 1
#define RIGHT_WEIGHT 1
#define UP_WEIGHT 1
#define DOWN_WEIGHT 1

//Data is as follows:
//0 - size of each copy of data
//Node data (all are pairs of x, y):
//PWR
//GND
//INPUTS
//OUTPUTS
//TERMINALS - Always D-G-S
__constant__ short node_layout[32000];

__global__ void rand_setup_kernel(hiprandState *state) {
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    hiprand_init(7+id, id, 0, &state[id]);
}

__global__ void ga_populate_kernel(hiprandState *state, short *pop_mem) {
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    hiprandState local_state = state[id];
    float rand_float;
    int rand;

    int mem_offset = id * node_layout[0];
    for (unsigned int i=0; i<node_layout[0]; ++i) {
	rand_float = hiprand_uniform(&local_state) * 100; //TODO Depend on num transistors
	rand = (int) rand_float;
	pop_mem[mem_offset+i] = rand;
    }
    //TODO Add contraints
    state[id] = local_state;
}

/*
__device__ float single_thread_fitness_func(short *layout, int mem_offset) {
    //Distance between connected terminals
    int offset = node_layout[4];
    int node_offset = 1 + offset + node_data[offset];
    int dist = 0;
    int size = 0;
    int angles = 0;
    short x[32]; //Will cause errors if any node has more that 32 terminals
    short y[32]; //Needed because cuda does not allow dynamically sized arrays
    short t, xdiff, ydiff;
    for (short i=0; i<node_data[offset]; ++i) {
        size = node_layout[offset+i+1];
	for (short j=0; j<size; ++j) {
	    t = node_layout[node_offset];
	    x[j] = layout[t-mem_offset];
	    y[j] = layout[t-mem_offset+1];
	    for (short k=0; k<j; ++k) {
		xdiff = x[j] - x[k];
		ydiff = y[j] - y[k];
		dist += xdiff * xdiff + ydiff * ydiff;

		//Angles
		angles += (xdiff == 0) + (ydiff == 0) - 1;
	    }
	    ++node_offset;
	}
    }
    angles = angles * ANGLE_WEIGHT;
    dist = dist * DISTANCE_WEIGHT;
    int left = 0;
    int right = 0;
    int up = 0;
    int down = 0;
    int pos = 0;
    //Inputs to left side
    for (short i=0; i<node_layout[2]; ++i) {
	pos = layout[i+4];
	left += pos * pos * LEFT_WEIGHT;
    }
    //Outputs to right side
    for (short i=0; i<node_layout[3]; ++i) {
	pos = layout[i+node_layout[2]*2+4] - 10; //TODO Change depend on num transistors
	right += pos * pos * RIGHT_WEIGHT;
    }
    //Power to top
    pos = layout[1];
    up = pos * pos * UP_WEIGHT;
    //Ground to bottom
    pos = layout[3] - 10;
    down = pos * pos * DOWN_WEIGHT;
    printf("Fitness function for thread %d output: %d, %d, %d, %d, %d", dist, left, right, up, down);
    return dist;
    }*/

__device__ float single_thread_fitness_func_mem(short* pop_mem, int mem_offset, int id) {
    //Distance between connected terminals
    int offset = node_layout[4];
    int node_offset = 1 + offset + node_layout[offset];
    int dist = 0;
    int size = 0;
    int angles = 0;
    short x[32]; //Will cause errors if any node has more that 32 terminals
    short y[32]; //Needed because cuda does not allow dynamically sized arrays
    short t, xdiff, ydiff;
    for (short i=0; i<node_layout[offset]; ++i) {
        size = node_layout[offset+i+1];
	for (short j=0; j<size; ++j) {
	    t = node_layout[node_offset];
	    x[j] = pop_mem[mem_offset+t];
	    y[j] = pop_mem[mem_offset+t+1];
	    for (short k=0; k<j; ++k) {
		xdiff = x[j] - x[k];
		ydiff = y[j] - y[k];
		dist += xdiff * xdiff + ydiff * ydiff;

		//Angles
		angles += (xdiff == 0) + (ydiff == 0) - 1;
	    }
	    ++node_offset;
	}
    }
    angles = angles * ANGLE_WEIGHT;
    dist = dist * DISTANCE_WEIGHT;
    int left = 0;
    int right = 0;
    int up = 0;
    int down = 0;
    int pos = 0;
    //Inputs to left side
    for (short i=0; i<node_layout[2]; ++i) {
	pos = pop_mem[mem_offset+i+4];
	left += pos * pos * LEFT_WEIGHT;
    }
    //Outputs to right side
    for (short i=0; i<node_layout[3]; ++i) {
	pos = pop_mem[mem_offset+i+node_layout[2]*2+4] - 10; //TODO Change depend on num transistors
	right += pos * pos * RIGHT_WEIGHT;
    }
    //Power to top
    pos = pop_mem[mem_offset+1];
    up = pos * pos * UP_WEIGHT;
    //Ground to bottom
    pos = pop_mem[mem_offset+3] - 10;
    down = pos * pos * DOWN_WEIGHT;
    printf("Fitness function for thread %d output: %d, %d, %d, %d, %d", dist, left, right, up, down);
    return dist;
}

__global__ void ga_fitness_kernel(short *pop_mem, float *fit_mem) {
    __shared__ float fit_scores[TILE_WIDTH];
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    int mem_offset = id * node_layout[0];
    //Makes local copy, might be faster to not copy
/*
    short layout[size];
    for (int i=0; i<node_layout[0]; ++i) {
	layout[i] = pop_mem[mem_offset + i];
    }
    fit_scores[threadIdx.x] = single_thread_fitness_func(layout, mem_offset);
*/
    fit_scores[threadIdx.x] = single_thread_fitness_func_mem(pop_mem, mem_offset, id);
}

int term_pos(Terminal *term, int offset_data, int offset_in, int offset_out) {
    switch(term->type) {
    case 'D':
	return 2 + offset_in + offset_out + (term->num - 1) * 3; //Order D-G-S
    case 'G':
	return 3 + offset_in + offset_out + (term->num - 1) * 3;
    case 'S':
	return 4 + offset_in + offset_out + (term->num - 1) * 3;
    case 'P':
	return 0; //Always first
    case 'Z':
	return 1; //Always second
    case 'I':
	return 1 + term->num; //Directly follows ground, term num starts at 1, so only offset 1
    case 'O':
	return 1 + offset_in + term->num; // Follows inputs
    default:
	cout << "Improper terminal" << endl;
	return -1;
    }
}

void create_node_data_array(Layout *main_layout, short *node_data, int offset_data, int offset_in, int offset_out) {
    Node* node;
    node_data[offset_data] = main_layout->nodes_size();
    int node_offset = 1 + offset_data + main_layout->nodes_size();
    for (unsigned int i=0; i<main_layout->nodes_size(); ++i) {
	node = main_layout->get_node(i);
	node_data[i+1+offset_data] = node->terms.size();
	for (unsigned int j=0; j<node->terms.size(); ++j) {
	    node_data[node_offset] = term_pos(node->terms[j], offset_data, offset_in, offset_out);
	    ++node_offset;
	}
    }
}

void launch_ga(Layout *main_layout) {
    //Specify block size
    const dim3 block_size(TILE_WIDTH);
    //Assume POP_SIZE is multiple of block size
    const dim3 num_blocks(POP_SIZE/block_size.x);

    int num_terminals = (main_layout->trans_size()) * 3;
    int offset_in = main_layout->in_size();
    int offset_out = main_layout->out_size();
    int trans_offset = offset_in + offset_out + 2;
    short per_copy_size = (trans_offset + num_terminals) * 2 * sizeof(short);
    int thread_count = num_blocks.x * num_blocks.y * block_size.x * block_size.y;
    int total_mem = thread_count * per_copy_size;

    cout << "Read in Layout. " << endl;
    cout << "Offset: " << trans_offset << endl;
    cout << "Number of transistor terminals: " << num_terminals << endl;


    cout << "\n\nBlock Allocation:\nDefined Tile size: " << TILE_WIDTH << endl;
    cout << "Defined population size: " << POP_SIZE << endl;
    cout << "Number of blocks: " << num_blocks.x * num_blocks.y << endl;
    cout << "Number of threads per block: " << block_size.x * block_size.y<< endl;
    cout << "Total number of threads: " << thread_count << endl;

    //Allocate memory
    cout << "\n\nMemory:\nSize of short: " << sizeof(short) << " bytes" << endl;
    cout << "Size of layout copy: " << per_copy_size << " bytes" << endl;
    cout << "Total layout memory use: " << total_mem << " bytes" << endl;

    int size = 0;
    Node* node;
    for (unsigned int i=0; i<main_layout->nodes_size(); ++i) {
	node = main_layout->get_node(i);
	size += node->terms.size();
    }

    int offset_data = 5;
    int node_mem_size = 1 + main_layout->nodes_size() + size + offset_data;
    short node_data[node_mem_size];

    node_data[0] = (trans_offset + num_terminals) * 2;
    node_data[1] = per_copy_size;
    node_data[2] = offset_in;
    node_data[3] = offset_out;
    node_data[4] = offset_data;

    cout << "Node size: " << node_mem_size << " numbers" << endl;
    cout << "Node memory use: " << node_mem_size * sizeof(short) << " bytes" << endl;

    create_node_data_array(main_layout, node_data, offset_data, offset_in, offset_out);

    //Random number generation setup
    hiprandState *rand_states = 0;
    hipMalloc((void **)&rand_states, thread_count * sizeof(hiprandState));
    rand_setup_kernel<<<num_blocks, block_size>>>(rand_states);

    short *pop_mem = 0;
    //short *node_mem = 0;
    float *fit_mem = 0;
    short *run_num = 0;

    hipMalloc((void**)&pop_mem, total_mem);
    //hipMalloc((void**)&node_mem, node_mem_size * sizeof(short));
    hipMalloc((void**)&fit_mem, thread_count * sizeof(float));
    hipMalloc((void**)&run_num, sizeof(short));

    //hipMemcpy(node_mem, node_data, node_mem_size * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(node_layout), node_data, node_mem_size * sizeof(short));
    hipMemset(run_num, 0, sizeof(short));

    ga_populate_kernel<<<num_blocks, block_size>>>(rand_states, pop_mem);

    ga_fitness_kernel<<<num_blocks, block_size>>>(pop_mem, fit_mem);

    short *host_pop = 0;
    host_pop = (short*)malloc(total_mem);

    hipMemcpy(host_pop, pop_mem, total_mem, hipMemcpyDeviceToHost);


    free(host_pop);
    
    hipFree(pop_mem);
    hipFree(fit_mem);
    hipFree(run_num);
}

